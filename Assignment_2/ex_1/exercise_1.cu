
#include <hip/hip_runtime.h>
#include <stdio.h>

#define N 256
#define TPB 256

__global__ void cuda_hello(){
    printf("Hello World! My threadId is %d\n", threadIdx.x);
}

int main() {
    /*
    Set the kernel to run with 256 threads in 1 single thread block,
    following a 1D distribution.
    */
    cuda_hello<<<N/TPB,TPB>>>();
    hipDeviceSynchronize();
    return 0;
}